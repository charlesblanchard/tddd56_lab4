#include "hip/hip_runtime.h"
// Matrix addition, GPU version
// nvcc matrix_gpu.cu -L /usr/local/cuda/lib -lcudart -o matrix_gpu

#include <stdio.h>


const int blocksize = 16; 
const int N = 1024;
const int gridsize = N / blocksize;

__global__
void add_matrix(float *a, float *b, float *c, int N)
{
	int index_x = blockIdx.x * blockDim.x + threadIdx.x;
	int index_y = blockIdx.y * blockDim.y + threadIdx.y;
	
	int grid_width = gridDim.x * blockDim.x;
	
	int index = index_y * grid_width + index_x;
	
	c[index] = a[index] + b[index];
}

int main()
{
	
	
	float *a = new float[N*N];
	float *b = new float[N*N];
	float *c = new float[N*N];
	
	unsigned long size = N*N*sizeof(float);
	
	float *gpu_a;
	float *gpu_b;
	float *gpu_c;

	for (int i = 0; i < N; i++)
		for (int j = 0; j < N; j++)
		{
			a[i+j*N] = 10 + i;
			b[i+j*N] = (float)j / N;
		}
		
	hipEvent_t begin;
	hipEvent_t end;
	
	float elapsed;
	
	hipEventCreate(&begin);
	hipEventCreate(&end);
	
	hipMalloc( (void**)&gpu_a, size);
	hipMalloc( (void**)&gpu_b, size);
	hipMalloc( (void**)&gpu_c, size);
	
	
	// dim3 dimBlock( blockDim.x, blockDim.y, 1);
	dim3 dimBlock( blocksize,blocksize,1);
	
	// dim3 dimGrid( blockIdx.x, blockIdx.y );
	dim3 dimGrid(  gridsize, gridsize );
	
	hipMemcpy( gpu_a, a, size, hipMemcpyHostToDevice ); 
	hipMemcpy( gpu_b, b, size, hipMemcpyHostToDevice ); 
	
	hipEventRecord(begin, 0);
	
	add_matrix<<<dimGrid, dimBlock>>>(gpu_a,gpu_b,gpu_c,N);
	
	hipDeviceSynchronize();
	hipEventRecord(end, 0);
	
	
	
	hipMemcpy( c, gpu_c, size, hipMemcpyDeviceToHost ); 
	hipFree( gpu_a );
	hipFree( gpu_b );
	hipFree( gpu_c );
	
	hipEventSynchronize(end);
	
	hipEventElapsedTime(&elapsed, begin, end);

	/*
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
		{
			printf("%0.2f ", c[i+j*N]);
		}
		printf("\n");
	}
	printf("\n");*/
	printf("Blocksize = %i\tN = %i Time : %f\n",blocksize,N,elapsed*1000);
}
