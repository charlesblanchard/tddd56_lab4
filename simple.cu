#include "hip/hip_runtime.h"
// Simple CUDA example by Ingemar Ragnemalm 2009. Simplest possible?
// Assigns every element in an array with its index.

// nvcc simple.cu -L /usr/local/cuda/lib -lcudart -o simple

#include <stdio.h>
#include <math.h>

const int N = 16; 
const int blocksize = 16; 

__global__ 
void simple(float *c) 
{
	//c[threadIdx.x] = threadIdx.x;
	c[threadIdx.x] = sqrt( c[threadIdx.x] );
}

int main()
{
	float *c = new float[N];
		
	float *cd;
	const int size = N*sizeof(float);
	
	for(int i=0; i<N; i++)
		c[i]=i;
	
	hipMalloc( (void**)&cd, size );
	dim3 dimBlock( blocksize, 1 );
	dim3 dimGrid( 1, 1 );
	
	hipMemcpy( cd, c, size, hipMemcpyHostToDevice ); 
	
	simple<<<dimGrid, dimBlock>>>(cd);
	hipDeviceSynchronize();
	hipMemcpy( c, cd, size, hipMemcpyDeviceToHost ); 
	hipFree( cd );
	
	printf("\n");
	for (int i = 0; i < N; i++)
		printf("Square root of %2i: CPU = %f\tGPU = %f \n",i, c[i], sqrt(i));
	printf("\n");
	delete[] c;
	printf("done\n");
	return EXIT_SUCCESS;
}
