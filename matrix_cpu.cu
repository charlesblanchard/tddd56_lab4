#include "hip/hip_runtime.h"
// Matrix addition, GPU version
// nvcc matrix_gpu.cu -L /usr/local/cuda/lib -lcudart -o matrix_gpu

#include <stdio.h>




const int N = 16; 
const int blocksize = 16; 

__global__
void add_matrix(float *a, float *b, float *c, int N)
{
	int index;
	
	for (int i = 0; i < N; i++)
		for (int j = 0; j < N; j++)
		{
			index = i + j*N;
			c[index] = a[index] + b[index];
		}
}

int main()
{
	const int N = 16;

	float a[N*N];
	float b[N*N];
	float c[N*N];

	for (int i = 0; i < N; i++)
		for (int j = 0; j < N; j++)
		{
			a[i+j*N] = 10 + i;
			b[i+j*N] = (float)j / N;
		}
	
	hipMalloc( (void**)&gpu_a, size );
	hipMalloc( (void**)&gpu_b, size );
	hipMalloc( (void**)&gpu_c, size );
	
	dim3 dimBlock( blocksize, blocksize );
	dim3 dimGrid( 1, 1 );
	
	hipMemcpy( &gpu_a, a, N*N, hipMemcpyHostToDevice ); 
	hipMemcpy( &gpu_b, b, N*N, hipMemcpyHostToDevice ); 
	
	add_matrix<<<dimGrid, dimBlock>>>(&gpu_a,&gpu_b,&gpu_c);
	hipDeviceSynchronize();
	hipMemcpy( c, gpu_c, N*N, hipMemcpyDeviceToHost ); 
	hipFree( cd );

	
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
		{
			printf("%0.2f ", c[i+j*N]);
		}
		printf("\n");
	}
}
